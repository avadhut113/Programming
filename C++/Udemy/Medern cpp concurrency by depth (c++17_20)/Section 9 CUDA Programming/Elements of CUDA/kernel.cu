#include "hip/hip_runtime.h"
﻿#include "OrganizationTheradCUDA.cuh"
#include "UniqueIndexCalculation.cuh"
#include "SumArray.cuh"

__global__ void hello_cuda()
{
    printf("Hello CUDA world\n");
}

void elements_of_CUDA()
{
    int nx = 16, ny = 4;
    dim3 block(8, 2);
    dim3 grid(nx / block.x, ny / block.y);

    hello_cuda << <grid, block >> > ();
}

int main()
{
    //elements_of_CUDA();
    
    //OrganizingCUDAThreads();

    //UniqueIDXCal();

    SumArray();
    return 0;
}